#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__ void update_diagonal(float *d_v, float *d_x, float *d_norm)
{
    if (*d_x > 0) {
        *d_v = *d_x - *d_norm;
    }else{
        *d_v = *d_x + *d_norm;
    }
}

__global__ void update_beta(float *d_beta, float *d_dotpro)
{
    *d_beta = -2 / *d_dotpro;
}

/*
    Computes W = beta A V + alpha W
*/
//__global__ void gtSgemv(float alpha, float *d_beta, )

void house(float *d_v, float *d_x, float *d_beta, int len, int m, int n)
{
    printf("1\n");
    hipError_t cudaStat;
    printf("2\n");
    hipblasStatus_t stat;
    printf("3\n");
    hipblasHandle_t handle;
    printf("4\n");
    hipblasCreate(&handle);
    printf("5\n");
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    printf("6\n");
    float *d_norm, *d_dotpro;
    cudaStat = hipMalloc((void**)&d_norm, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&d_dotpro, sizeof(float) * 1);
    printf("point0\n");
    cudaStat = hipMemset(d_norm, 0, sizeof(float) * 1);
    cudaStat = hipMemset(d_dotpro, 0, sizeof(float) * 1);
    hipDeviceSynchronize();
    printf("point1\n");
//    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);  
    stat = hipblasSnrm2(handle, len, d_x, 1, d_norm);
    hipDeviceSynchronize();

    printf("point2\n");
//    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    stat = hipblasScopy(handle, len, d_x, 1, d_v, 1);
    hipDeviceSynchronize();
    update_diagonal<<<1, 1>>>(d_v, d_x, d_norm);
    hipDeviceSynchronize();
    printf("point3\n");
//    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    stat = hipblasSdot(handle, len, d_v, 1, d_v, 1, d_dotpro);
    hipDeviceSynchronize();
    printf("point4\n");
    update_beta<<<1, 1>>>(d_beta, d_dotpro);
    hipDeviceSynchronize();
    printf("point5\n");
    hipblasDestroy(handle);
    hipFree(d_norm);
    hipFree(d_dotpro);
    
    return;
}

void apply_house(float *d_v, float *d_A, float *d_beta, int len, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    int sub_m = len;
    int sub_n = n - m + len;
    
    float *d_v_A;
    printf("point1\n");
    cudaStat = hipMalloc((void**)&d_v_A, sizeof(float) * sub_n);
    printf("point2\n");
    cudaStat = hipMemset(d_v_A, 0, sizeof(float) * sub_n);

    float *alpha;
    float *beta;
    cudaStat = hipMalloc((void**)&alpha, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&beta, sizeof(float) * 1);
    cudaStat = hipMemset(alpha, 1, sizeof(float) * 1);
    cudaStat = hipMemset(beta, 0, sizeof(float) * 1);
    printf("point3\n");
    hipDeviceSynchronize();
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, sub_n, sub_m, alpha, d_v, sub_m, d_A, m, beta, d_v_A, 1);
    hipDeviceSynchronize();

    cudaStat = hipMemset(beta, 1, sizeof(float) * 1);
    printf("point4\n");
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, 1, d_beta, d_v, sub_m, d_v_A, 1, beta, d_A, sub_m);
    hipDeviceSynchronize();
    printf("point5\n");

    hipblasDestroy(handle);
    hipFree(d_v_A);
    
    return;
    
}

void qr_calculate(float *d_A, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    //hipblasHandle_t handle;
    
    int len = 0;
    float *d_beta, *d_house_v;
    cudaStat = hipMalloc((void**)&d_beta,sizeof(float)*1);
    cudaStat = hipMalloc((void**)&d_house_v,sizeof(float)*m);
    hipDeviceSynchronize();

    for (int k = 0; k < n; k++) {
        
        //householder reflector
        len = m - k;
        printf("%d access house()\n", k);
        house(d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n);
        
        //apply householder reflector
        printf("%d access apply_house()\n", k);
        apply_house(d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n);
    }
}

int main()
{
    int m = 4, n = 3, i;
    float A[m*n] = { 1, 1, 1, 1, -1, 4, 4, -1, 4, -2, 2, 0 };
    float *d_A;
    
    hipError_t cudaStat;
    cudaStat = hipMalloc((void**)&d_A,sizeof(float)*m*n);
    cudaStat = hipMemcpy(d_A, A, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    qr_calculate(d_A, m, n);
    
    cudaStat = hipMemcpy(A, d_A, sizeof(float)*m*n, hipMemcpyDeviceToHost);
    
    for (i = 0; i < m*n; i++) {
        printf("%f ", A[i]);
    }
    printf("\n");
    
    return 0;
}
