#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__ void update_diagonal(float *d_v, float *d_x, float *d_norm)
{
    if(threadIdx.x == 1)
    {
        if (*d_x > 0) {
            d_v[0] = *d_x - *d_norm;
        }else{
            d_v[0] = *d_x + *d_norm;
        }
    }
}

__global__ void update_beta(float *d_beta, float *d_dotpro)
{
    if(threadIdx.x == 1)
    {
        *d_beta = -2 / *d_dotpro;
    }
}

__global__ void mycopy(float *d_v, float *d_x, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < len)
    {
        d_v[tid] = d_x[tid];
    }

}

/*
    Computes W = beta A V + alpha W
*/
//__global__ void gtSgemv(float alpha, float *d_beta, )

void house(float *d_v, float *d_x, float *d_beta, int len, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    float *d_norm, *d_dotpro;
    float ZERO = 0;
    cudaStat = hipMalloc((void**)&d_norm, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&d_dotpro, sizeof(float) * 1);
    
    cudaStat = hipMemcpy(d_norm, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(d_dotpro, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    stat = hipblasSnrm2(handle, len, d_x, 1, d_norm);
    hipDeviceSynchronize();


    stat = hipblasScopy(handle, len, d_x, 1, d_v, 1);
//    mycopy<<<1024*16, 256>>>(d_v, d_x, len);

    hipDeviceSynchronize();
    update_diagonal<<<1, 8>>>(d_v, d_x, d_norm);
    hipDeviceSynchronize();

    stat = hipblasSdot(handle, len, d_v, 1, d_v, 1, d_dotpro);
    hipDeviceSynchronize();
    
    update_beta<<<1, 8>>>(d_beta, d_dotpro);
    hipDeviceSynchronize();
    
    hipblasDestroy(handle);
    hipFree(d_norm);
    hipFree(d_dotpro);
    
    return;
}

void apply_house(float *d_v, float *d_A, float *d_beta, int len, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int sub_m = len;
    int sub_n = n - m + len;
    float ONE = 1, ZERO = 0;
    
    float *d_v_A;
    cudaStat = hipMalloc((void**)&d_v_A, sizeof(float) * sub_n);
    cudaStat = hipMemset((void*)d_v_A, ZERO, sizeof(float) * sub_n);

    float *alpha;
    float *beta;
    cudaStat = hipMalloc((void**)&alpha, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&beta, sizeof(float) * 1);
    cudaStat = hipMemcpy(alpha, &ONE, sizeof(float) * 1, hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(beta, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    hipDeviceSynchronize();
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, sub_n, sub_m, alpha, d_v, 1, d_A, m, beta, d_v_A, 1);
    hipDeviceSynchronize();

    
    cudaStat = hipMemcpy(beta, &ONE, sizeof(float) * 1, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
   
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, 1, d_beta, d_v, sub_m, d_v_A, 1, beta, d_A, m);
    hipDeviceSynchronize();
   

    hipblasDestroy(handle);
    hipFree(d_v_A);
    
    return;
    
}

void qr_calculate(float *d_A, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
    
    float A[m*n];
    int len = 0;
    float *d_beta, *d_house_v;
    cudaStat = hipMalloc((void**)&d_beta,sizeof(float)*1);
    cudaStat = hipMalloc((void**)&d_house_v,sizeof(float)*m);
    hipDeviceSynchronize();

    for (int k = 0; k < n; k++) {
        
        //householder reflector
        len = m - k;
        printf("%d access house()\n", k);
        house(d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n);
        
        //apply householder reflector
        printf("%d access apply_house()\n", k);
        apply_house(d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n);
    }
}

int main()
{
    int m = 4, n = 3, i;
    float A[m*n] = { 1, 1, 1, 1, -1, 4, 4, -1, 4, -2, 2, 0 };
    float *d_A;
      
    hipError_t cudaStat;
    cudaStat = hipMalloc((void**)&d_A,sizeof(float)*m*n);
    cudaStat = hipMemcpy(d_A, A, sizeof(float)*m*n, hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
    
    qr_calculate(d_A, m, n);
    
    cudaStat = hipMemcpy(A, d_A, sizeof(float)*m*n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    for (i = 0; i < m*n; i++) {
        printf("%f ", A[i]);
    }
    printf("\n");
    
    return 0;
}
