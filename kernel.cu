#include "hip/hip_runtime.h"
//#include <stdio.h>
//#include <stdlib.h>
//#include <hip/hip_runtime.h>
//#include "hipblas.h"
#include "kernel.h"
#include <assert.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__ void update_diagonal(double *d_v, double *d_x, double *d_norm)
{ 
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid == 1)
    {
        if (*d_x >= 0) {
            d_v[0] = *d_x - *d_norm;
        }else{
            d_v[0] = *d_x + *d_norm;
        }
    }
}

__global__ void update_beta(double *d_beta, double *d_dotpro)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid == 1)
    {
        *d_beta = (double)(-2 / *d_dotpro);
    }
}

__global__ void mycopy(double *W, double *V, double *beta, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < len)
    {
        W[tid] = (*beta) * V[tid];
    }

}

__global__ void initial_float(double *in, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < len)
    {
        in[tid] = 0.0;
    }

//    if(tid * 4 + 4 <= len)
//    {
//        in[tid * 4] = 0.0f;
//        in[tid * 4 + 1] = 0.0f;
//        in[tid * 4 + 2] = 0.0f;
//        in[tid * 4 + 3] = 0.0f;
//        in[tid * 8 + 4] = 0.0f;
//        in[tid * 8 + 5] = 0.0f;
//        in[tid * 8 + 6] = 0.0f;
//        in[tid * 8 + 7] = 0.0f;
//    }
//    if(len % 4 == tid)
//    {
//        in[len - (len % 4) + tid] = 0.0f;
//    }
}

/*
Computes W = beta A V + alpha W
*/
//__global__ void gtSgemv(float alpha, float beta, float *A, float *V, int M, int N, float *W)


void house(hipblasHandle_t handle, double *d_v, double *d_x, double *d_beta, int len, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;

    double *d_norm, *d_dotpro;
    double ZERO = 0.0;
    
    cudaStat = hipMalloc((void**)&d_norm, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&d_dotpro, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);

    cudaStat = hipMemcpy(d_norm, &ZERO, sizeof(double) * 1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(d_dotpro, &ZERO, sizeof(double) * 1, hipMemcpyHostToDevice);  
    assert(hipSuccess == cudaStat);

    hipDeviceSynchronize();

    stat = hipblasDnrm2(handle, len, d_x, 1, d_norm);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    stat = hipblasDcopy(handle, len, d_x, 1, d_v, 1);
    //mycopy<<<1024*16, 256>>>(d_v, d_x, len);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    update_diagonal<<<1, 8>>>(d_v, d_x, d_norm);
    hipDeviceSynchronize();

    assert(hipGetLastError() == hipSuccess);

    stat = hipblasDdot(handle, len, d_v, 1, d_v, 1, d_dotpro);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    update_beta<<<1, 8>>>(d_beta, d_dotpro);
    hipDeviceSynchronize();

    assert(hipGetLastError() == hipSuccess);

    hipFree(d_norm);
    hipFree(d_dotpro);
    
    return;
}

void apply_house(hipblasHandle_t handle, double *d_v, double *d_A, double *d_beta, int len, int m, int n, int r)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    
    int sub_m = len;
    int sub_n;
    if (r == 1) {
        sub_n = n - m + len;
    }else{
        sub_n = (len % r == 0 ? r : (len % r));
    }
    double ONE = 1.0, ZERO = 0.0;

    double *d_v_A;
    cudaStat = hipMalloc((void**)&d_v_A, sizeof(double) * sub_n);
    //cudaStat = hipMemset((void*)d_v_A, ZERO, sizeof(float) * sub_n);
    assert(hipSuccess == cudaStat);
 
    initial_float<<<1024*64, 1024>>>(d_v_A, sub_n);
    hipDeviceSynchronize();

    assert(hipGetLastError() == hipSuccess);

    double *alpha;
    double *beta;
    cudaStat = hipMalloc((void**)&alpha, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&beta, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(alpha, &ONE, sizeof(double) * 1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(beta, &ZERO, sizeof(double) * 1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);
    hipDeviceSynchronize();

    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, sub_n, sub_m, alpha, d_v, 1, d_A, m, beta, d_v_A, 1);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    cudaStat = hipMemcpy(beta, &ONE, sizeof(double) * 1, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    assert(hipSuccess == cudaStat);

    stat = hipblasDger(handle, sub_m, sub_n, d_beta, d_v, 1, d_v_A, 1, d_A, m);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    hipFree(d_v_A);
    hipFree(alpha);
    hipFree(beta);    
    return;

}

void generate_WY(hipblasHandle_t handle, double *W, const double *Y, double *d_beta, int m, int n, int len, int r)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
   
    double ONE = 1.0, ZERO = 0.0;
    double *d_Y_v;
    double *alpha;
    double *beta;
    cudaStat = hipMalloc((void**)&d_Y_v, sizeof(double) * r);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&alpha, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&beta, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(alpha, &ONE, sizeof(double) * 1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(beta, &ZERO, sizeof(double) * 1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);
    hipDeviceSynchronize();

    initial_float<<<1024*64, 1024>>>(d_Y_v, r);
    hipDeviceSynchronize();

    assert(hipGetLastError() == hipSuccess);

    //mycopy<<<1024*16, 256>>>(Y, W, d_beta, len);
    stat = hipblasDaxpy(handle, len, d_beta, Y, 1, W, 1);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);
    //stat = hipblasSaxpy(handle, len*(r-1), alpha, &Y[IDX2C(0,1,len)], 1, &W[IDX2C(0,1,len)], 1);

    stat = hipblasDcopy(handle, len*(r-1), &Y[IDX2C(0,1,len)], 1, &W[IDX2C(0,1,len)], 1);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    for (int j = 1; j < r; j++) {
        stat = hipblasDgemv(handle, HIPBLAS_OP_T, len, j, alpha, Y, len, &Y[IDX2C(0,j,len)], 1, beta, d_Y_v, 1);
        hipDeviceSynchronize();
        assert(stat == HIPBLAS_STATUS_SUCCESS);

        stat = hipblasDgemv(handle, HIPBLAS_OP_N, len, j, &d_beta[j], W, len, d_Y_v, 1, &d_beta[j], &W[IDX2C(0,j,len)], 1);
        hipDeviceSynchronize();
        assert(stat == HIPBLAS_STATUS_SUCCESS);

    }

    hipFree(d_Y_v);
    hipFree(alpha);
    hipFree(beta);

    return;
}

void apply_WY(hipblasHandle_t handle, double *d_A, double *W, double *Y, int m, int n, int len, int r)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    
    double *d_W_A;
    double *alpha;
    double *beta;
    double ONE = 1.0, ZERO = 0.0;
    int sub_m = len;
    int sub_n = n - m + len;
    cudaStat = hipMalloc((void**)&d_W_A, sizeof(double) * r * sub_n);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&alpha, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&beta, sizeof(double) * 1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(alpha, &ONE, sizeof(double) * 1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(beta, &ZERO, sizeof(double) * 1, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);

    initial_float<<<1024*64, 1024>>>(d_W_A, r*sub_n);
    hipDeviceSynchronize();

    assert(hipGetLastError() == hipSuccess);

    hipDeviceSynchronize();
    stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, r, sub_n, sub_m, alpha, W, sub_m, d_A, m, beta, d_W_A, r);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, r, alpha, Y, sub_m, d_W_A, r, alpha, d_A, m);
    hipDeviceSynchronize();
    assert(stat == HIPBLAS_STATUS_SUCCESS);

    hipFree(d_W_A);
    hipFree(alpha);
    hipFree(beta);

    return;
}

void unblocked_qr_calculate(double *d_A, int m, int n)
{
    hipError_t cudaStat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    int len = 0;
    double *d_beta, *d_house_v;
    cudaStat = hipMalloc((void**)&d_beta,sizeof(double)*1);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&d_house_v,sizeof(double)*m);
    assert(hipSuccess == cudaStat);
    hipDeviceSynchronize();

    initial_float<<<1024*64, 1024>>>(d_house_v, m);
    hipDeviceSynchronize();

    assert(hipGetLastError() == hipSuccess);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0, housetime = 0, applytime = 0;

    for (int k = 0; k < n; k++) {

        //householder reflector
        len = m - k;
      
        hipEventRecord(start);

        //printf("%d access house()\n", k);
        house(handle, d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        housetime += milliseconds;

        hipEventRecord(start);

        //apply householder reflector
        //printf("%d access apply_house()\n", k);
        apply_house(handle, d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n, 1);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        applytime += milliseconds;

    }
    printf("house: %f \n", housetime);
    printf("apply house: %f \n", applytime);

    hipFree(d_beta);
    hipFree(d_house_v);
    return;
}

void blocked_qr_calculate(double *d_A, int m, int n, int r)
{
    if (n % r != 0) {
        return;
    }
    int num_block = n / r;
    hipError_t cudaStat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    int len = 0, sub_len;
    int first_row_ind = 0, ind;
    double *d_beta, *d_house_v;
    double *W;
    
    cudaStat = hipMalloc((void**)&d_beta,sizeof(double)*r);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&d_house_v,sizeof(double)*m*r);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMalloc((void**)&W,sizeof(double)*m*r);
    assert(hipSuccess == cudaStat);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0, housetime = 0, applytime = 0, WYtime = 0, applyWYtime = 0;

    for (int k = 0; k < num_block; k++) {
        first_row_ind = k * r;
        len = m - first_row_ind;
        initial_float<<<1024*64, 1024>>>(d_beta, r);
        hipDeviceSynchronize();
        assert(hipGetLastError() == hipSuccess);
        initial_float<<<1024*64, 1024>>>(d_house_v, len*r);
        hipDeviceSynchronize();
        assert(hipGetLastError() == hipSuccess);
        initial_float<<<1024*64, 1024>>>(W, len*r);
        hipDeviceSynchronize();
        assert(hipGetLastError() == hipSuccess);

        for (int j = 0; j < r; j++) {
            ind = first_row_ind + j;
            sub_len = len - j;

            hipEventRecord(start);

            //printf("block %d, row %d, access house()\n", k, j);
            //householder reflector
            house(handle, &d_house_v[IDX2C(j,j,len)], &d_A[IDX2C(ind,ind,m)], &d_beta[j], sub_len, m, n);
            
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            housetime += milliseconds;

            hipEventRecord(start);

            //printf("block %d, row %d, access apply_house()\n", k, j);
            //apply householder reflector
            apply_house(handle, &d_house_v[IDX2C(j,j,len)], &d_A[IDX2C(ind,ind,m)], &d_beta[j], sub_len, m, n, r);
 
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            applytime += milliseconds;

        }
        if(len == m - n + r)
            break;

        hipEventRecord(start);

        //printf("block %d, access grenerate_WY()\n", k);
        generate_WY(handle, W, d_house_v, d_beta, m, n, len, r);
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        WYtime += milliseconds;

        hipEventRecord(start);

        //apply W & Y
        //printf("block %d, access apply_WY()\n", k);
        apply_WY(handle, &d_A[IDX2C(first_row_ind,first_row_ind+r,m)], W, d_house_v, m, n, len, r);
    
        hipEventRecord(stop);         
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        applyWYtime += milliseconds;

    }

    printf("house: %f \n", housetime);
    printf("apply house: %f \n", applytime);
    printf("WY: %f \n", WYtime);
    printf("apply WY: %f \n", applyWYtime);

    hipFree(W);
    hipFree(d_house_v);
    hipFree(d_beta);
    return;
}
