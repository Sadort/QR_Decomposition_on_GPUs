#include "hip/hip_runtime.h"
//#include <stdio.h>
//#include <stdlib.h>
//#include <hip/hip_runtime.h>
//#include "hipblas.h"
#include "kernel.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

__global__ void update_diagonal(float *d_v, float *d_x, float *d_norm)
{
    if(threadIdx.x == 1)
    {
        if (*d_x >= 0) {
            d_v[0] = *d_x - *d_norm;
        }else{
            d_v[0] = *d_x + *d_norm;
        }
    }
}

__global__ void update_beta(float *d_beta, float *d_dotpro)
{
    if(threadIdx.x == 1)
    {
        *d_beta = (float)(-2 / *d_dotpro);
    }
}

__global__ void mycopy(float *W, float *V, float *beta, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < len)
    {
        W[tid] = (*beta) * V[tid];
    }

}

__global__ void initial_float(float *in, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < len)
    {
        in[tid] = 0.0f;
    }
}

/*
Computes W = beta A V + alpha W
*/
//__global__ void gtSgemv(float alpha, float beta, float *A, float *V, int M, int N, float *W)


void house(hipblasHandle_t handle, float *d_v, float *d_x, float *d_beta, int len, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;

    float *d_norm, *d_dotpro;
    float ZERO = 0.0f;
    
    cudaStat = hipMalloc((void**)&d_norm, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&d_dotpro, sizeof(float) * 1);
  
    cudaStat = hipMemcpy(d_norm, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(d_dotpro, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);  

    hipDeviceSynchronize();

    stat = hipblasSnrm2(handle, len, d_x, 1, d_norm);
    hipDeviceSynchronize();

    stat = hipblasScopy(handle, len, d_x, 1, d_v, 1);
    //mycopy<<<1024*16, 256>>>(d_v, d_x, len);

    hipDeviceSynchronize();
    update_diagonal<<<1, 8>>>(d_v, d_x, d_norm);
    hipDeviceSynchronize();
    
    stat = hipblasSdot(handle, len, d_v, 1, d_v, 1, d_dotpro);
    hipDeviceSynchronize();
    
    update_beta<<<1, 8>>>(d_beta, d_dotpro);
    hipDeviceSynchronize();

    hipFree(d_norm);
    hipFree(d_dotpro);
    
    return;
}

void apply_house(hipblasHandle_t handle, float *d_v, float *d_A, float *d_beta, int len, int m, int n, int r)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    
    int sub_m = len;
    int sub_n;
    if (r == 1) {
        sub_n = n - m + len;
    }else{
        sub_n = (len % r == 0 ? r : (len % r));
    }
    float ONE = 1.0f, ZERO = 0.0f;

    float *d_v_A;
    cudaStat = hipMalloc((void**)&d_v_A, sizeof(float) * sub_n);
    //cudaStat = hipMemset((void*)d_v_A, ZERO, sizeof(float) * sub_n);

    float *alpha;
    float *beta;
    cudaStat = hipMalloc((void**)&alpha, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&beta, sizeof(float) * 1);
    cudaStat = hipMemcpy(alpha, &ONE, sizeof(float) * 1, hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(beta, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    
    hipDeviceSynchronize();
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, sub_n, sub_m, alpha, d_v, 1, d_A, m, beta, d_v_A, 1);
    hipDeviceSynchronize();
    

    cudaStat = hipMemcpy(beta, &ONE, sizeof(float) * 1, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    
    stat = hipblasSger(handle, sub_m, sub_n, d_beta, d_v, 1, d_v_A, 1, d_A, m);
    hipDeviceSynchronize();
    
    hipFree(d_v_A);
    
    return;

}

void generate_WY(hipblasHandle_t handle, float *W, const float *Y, float *d_beta, int m, int n, int len, int r)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
   
    float ONE = 1.0f, ZERO = 0.0f;
    float *d_Y_v;
    float *alpha;
    float *beta;
    cudaStat = hipMalloc((void**)&d_Y_v, sizeof(float) * r);
    cudaStat = hipMalloc((void**)&alpha, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&beta, sizeof(float) * 1);
    cudaStat = hipMemcpy(alpha, &ONE, sizeof(float) * 1, hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(beta, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);

    initial_float<<<1024*16, 256>>>(d_Y_v, r);
    hipDeviceSynchronize();
    
    //mycopy<<<1024*16, 256>>>(Y, W, d_beta, len);
    stat = hipblasSaxpy(handle, len, d_beta, Y, 1, W, 1);
    stat = hipblasSaxpy(handle, len*(r-1), alpha, &Y[IDX2C(0,1,len)], 1, &W[IDX2C(0,1,len)], 1);
    hipDeviceSynchronize();
    
    for (int j = 1; j < r; j++) {
        stat = hipblasSgemv(handle, HIPBLAS_OP_T, len, j, alpha, Y, len, &Y[IDX2C(0,j,len)], 1, beta, d_Y_v, 1);
        hipDeviceSynchronize();

        stat = hipblasSgemv(handle, HIPBLAS_OP_N, len, j, &d_beta[j], W, len, d_Y_v, 1, &d_beta[j], &W[IDX2C(0,j,len)], 1);
        hipDeviceSynchronize();
    }

    hipFree(d_Y_v);
    hipFree(alpha);
    hipFree(beta);

    return;
}

void apply_WY(hipblasHandle_t handle, float *d_A, float *W, float *Y, int m, int n, int len, int r)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    
    float *d_W_A;
    float *alpha;
    float *beta;
    float ONE = 1.0f, ZERO = 0.0f;
    int sub_m = len;
    int sub_n = n - m + len;
    cudaStat = hipMalloc((void**)&d_W_A, sizeof(float) * r * sub_n);
    cudaStat = hipMalloc((void**)&alpha, sizeof(float) * 1);
    cudaStat = hipMalloc((void**)&beta, sizeof(float) * 1);
    cudaStat = hipMemcpy(alpha, &ONE, sizeof(float) * 1, hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(beta, &ZERO, sizeof(float) * 1, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, r, sub_n, sub_m, alpha, W, sub_m, d_A, m, beta, d_W_A, r);
    hipDeviceSynchronize();

    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, sub_m, sub_n, r, alpha, Y, sub_m, d_W_A, r, alpha, d_A, m);
    hipDeviceSynchronize();


    hipFree(d_W_A);
    hipFree(alpha);
    hipFree(beta);

    return;
}

void unblocked_qr_calculate(float *d_A, int m, int n)
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    int len = 0;
    float *d_beta, *d_house_v;
    cudaStat = hipMalloc((void**)&d_beta,sizeof(float)*1);
    cudaStat = hipMalloc((void**)&d_house_v,sizeof(float)*m);
    hipDeviceSynchronize();

    for (int k = 0; k < n; k++) {

        //householder reflector
        len = m - k;
        printf("%d access house()\n", k);
        house(handle, d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n);

        //apply householder reflector
        printf("%d access apply_house()\n", k);
        apply_house(handle, d_house_v, &d_A[IDX2C(k,k,m)], d_beta, len, m, n, 1);
    }
}

void blocked_qr_calculate(float *d_A, int m, int n, int r)
{
    if (n % r != 0) {
        return;
    }
    int num_block = n / r;
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

    int len = 0, sub_len;
    float ZERO = 0.0f;
    int first_row_ind = 0, ind;
    float *d_beta, *d_house_v;
    float *W, *Y;
    
    cudaStat = hipMalloc((void**)&d_beta,sizeof(float)*r);
    cudaStat = hipMalloc((void**)&d_house_v,sizeof(float)*m*r);
    cudaStat = hipMalloc((void**)&W,sizeof(float)*m*r);
    hipDeviceSynchronize();


    for (int k = 0; k < num_block; k++) {
        first_row_ind = k * r;
        len = m - first_row_ind;
        initial_float<<<1024*16, 256>>>(d_beta, r);
        initial_float<<<1024*16, 256>>>(d_house_v, len*r);
        initial_float<<<1024*16, 256>>>(W, len*r);

        hipDeviceSynchronize();

        for (int j = 0; j < r; j++) {
            ind = first_row_ind + j;
            sub_len = len - j;
            printf("block %d, row %d, access house()\n", k, j);
            //householder reflector
            house(handle, &d_house_v[IDX2C(j,j,len)], &d_A[IDX2C(ind,ind,m)], &d_beta[j], sub_len, m, n);
            
            printf("block %d, row %d, access apply_house()\n", k, j);
            //apply householder reflector
            apply_house(handle, &d_house_v[IDX2C(j,j,len)], &d_A[IDX2C(ind,ind,m)], &d_beta[j], sub_len, m, n, r);
           
        }
        if(len == m - n + r)
            return;

        printf("block %d, access grenerate_WY()\n", k);
        generate_WY(handle, W, d_house_v, d_beta, m, n, len, r);
        
        //apply W & Y
        printf("block %d, access apply_WY()\n", k);
        apply_WY(handle, &d_A[IDX2C(first_row_ind,first_row_ind+r,m)], W, d_house_v, m, n, len, r);
        
    }
}
